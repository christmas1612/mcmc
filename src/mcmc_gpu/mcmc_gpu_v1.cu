#include "hip/hip_runtime.h"
/*
 * -Implementation of MCMC Metropolis-Hastings Algorithm
 *  using GPU for processing
 * - Likelihood Kernel:
 *    performs dot product and exp on all data points 
 *    each block represents a data point 
 *      -> max dim=1024
 *      -> blocksPerGrid = Nd
 *    returns array of L_n (size of Nd)
 */
extern "C" {
#include "mcmc_gpu.h"
}

const int PRIOR_SD = 10;

__global__ void Likelihood_v1_sequential_addressing(double *samples, double *data, int8_t *labels,
                                                      double *L_n)
{
  __shared__ double shared_mem[];
  int data_idx = blockIdx.x * blockDim.x + threadIdx.x; // linearised datapoint space
  int label_idx = blockIdx.x;   // one datapoint exists per block
  int tidx = threadIdx.x;

  // calculates and loads a single dot product to shared memory
  shared_mem[tidx] = -labels[label_idx] * data[data_idx] * samples[tidx];
  __syncthreads();

  // perform reduction using shared memory
  int i = blockDim.x/2;
  while (i != 0) {
    if (tidx < i)
      shared_mem[tidx] += shared_mem[tidx + i];
    __syncthreads();
    i /= 2; 
  }
  // write result for this block back to global mem
  if (tidx == 0) 
    L_n[blockIdx.x] = -log(1 + exp(shared_mem[0])); 
}

void gpu_sampler(data_str data, gsl_rng *r, mcmc_str mcin,
                  mcmc_tune_str mct, mcmc_v_str mcdata,
                  out_str *out_par)
{
  print_gpu_info();
  hipSetDevice(0);

  mcmc_int_v mclocv;
  mcmc_int mcloc;
  malloc_mcmc_vectors_gpu(&mclocv, mcin);
  
  // set up the gpu vectors
  sz_str sz;
  sz.samples_map = mcin.dmap * sizeof(double);
  sz.samples_actual = mcin.ddata * sizeof(double);
  sz.data = mcin.dmap * mcin.Ndmap * sizeof(double);
  sz.labels = mcin.Ndmap * sizeof(int8_t);
  sz.likelihood = mcin.Ndmap * sizeof(double);

  double *host_lhood = (double *) malloc(sz.likelihood * sizeof(double));
  double *dev_samples, *dev_data, *dev_lhood;
  int8_t *dev_labels;
  hipMalloc(&dev_samples, sz.samples_map);
  hipMalloc(&dev_data, sz.data);
  hipMalloc(&dev_labels, sz.labels);
  hipMalloc(&dev_lhood, sz.likelihood);    // kernel will return a vector of likelihoods  

  // initialize likelihood to zeros (zero padding)
  memeset(host_lhood, 0, sz.likelihood * sizeof(double));
  // load data, labels, zero padded samples and likelihood on GPU
  hipMemcpy(dev_data, data.gpudata, sz.data, hipMemcpyHostToDevice);
  hipMemcpy(dev_labels, data.gpulabels, sz.labels, hipMemcpyHostToDevice);
  hipMemcpy(dev_samples, mclocv.current, sz.samples_map, hipMemcpyHostToDevice);
  hipMemcpy(dev_lhood, host_lhood, sz.likelihood, hipMemcpyHostToDevice);  

  // tune_target_a_gpu(r, mcin, &mct, mclocv, mcloc, sz, 
  //                     dev_samples, dev_data, dev_labels,
  //                     dev_lhood, host_lhood);
  tune_target_a_gpu_v2(r, mcin, &mct, mclocv, mcloc, sz, 
                        dev_samples, dev_data, dev_labels,
                        dev_lhood, host_lhood);
  // tune_ess_gpu(r, mcin, &mct, mclocv, mcloc, sz, 
  //               dev_samples, dev_data, dev_labels,
  //               dev_lhood, host_lhood);

  burn_in_metropolis_gpu(r, mcin, mct, mcdata, mclocv, &mcloc, sz,
                          dev_samples, dev_data, dev_labels,
                          dev_lhood, host_lhood);
  
  int accepted_samples = 0;
  clock_t start, stop;  

  start  = clock();
  metropolis_gpu(r, mcin, mct, mcdata, mclocv, &mcloc, &accepted_samples, sz,
                  dev_samples, dev_data, dev_labels, dev_lhood, host_lhood);
  stop = clock() - start;
  
  out_par->time_m = stop / (CLOCKS_PER_SEC * 60);
  out_par->time_s = (stop / CLOCKS_PER_SEC) - (out_par->time_m * 60);
  out_par->time_ms = (stop * 1000 / CLOCKS_PER_SEC) - (out_par->time_s * 1000) 
                      - (out_par->time_m * 1000 * 60);
  out_par->acc_ratio = (double)accepted_samples / mcin.Ns;

  free_mcmc_vectors_gpu(mclocv);
}

void metropolis_gpu(gsl_rng *r, mcmc_str mcin,
                    mcmc_tune_str mct, mcmc_v_str mcdata, mcmc_int_v mclocv, 
                    mcmc_int *mcloc, int *accepted_samples, sz_str sz,
                    double *dev_samples, double *dev_data, double *dev_labels,
                    double *dev_lhood, double *host_lhood)
{
  int i, dim_idx;
  int lhood_idx;
  double plhood;

  fprintf(stdout, "Starting metropolis algorithm. Selected rwsd = %f\n", mct.rwsd); 
  
  for(i=0; i<mcin.Ns; i++)
  {
    // propose next sample (note that from mcin.ddata -> mcin.dmap is zero padded)
    for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] + gsl_ran_gaussian_ziggurat(r, mct.rwsd); // random walk using Marsaglia-Tsang ziggurat algorithm
    
    // load proposed samples on GPU
    hipMemcpy(dev_samples, mclocv.proposed, sz.samples_actual, hipMemcpyHostToDevice);
    // load kernel, calculate the proposed likelihood
    Likelihood_v1_sequential_addressing<<<mcin.Nd, mcin.dmap>>>>(dev_samples, dev_data, dev_labels, 
                                                                 dev_lhood);
    // return back a vector of likelihoods
    hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
    // finish single result on CPU
    plhood = 0;
    for(lhood_idx = 0; lhood_idx < mcin.Nd; lhood_idx++){
      plhood += host_lhood[lhood_idx];
    }

    // calculate acceptance ratio
    mcloc->acceptance = acceptance_ratio(mclocv, mcloc, mcin, plhood);
    
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)
    {
      // accept proposed sample
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
      {
        mcdata.samples[i*mcin.ddata + dim_idx] = mclocv.proposed[dim_idx];
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior;
      *accepted_samples += 1;
    }else{
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
        mcdata.samples[i*mcin.ddata + dim_idx] = mclocv.current[dim_idx];
    }    
  } 
  fprintf(stdout, "Metropolis algorithm finished. Accepted Samples = %d\n\n", *accepted_samples);
}

void burn_in_metropolis_gpu(gsl_rng *r, mcmc_str mcin, mcmc_tune_str mct, mcmc_v_str mcdata, 
                            mcmc_int_v mclocv, mcmc_int *mcloc, sz_str sz,
                            double *dev_samples, double *dev_data, double *dev_labels,
                            double *dev_lhood, double *host_lhood)
{
  int i, dim_idx, lhood_idx;
  double plhood, clhood;
  fprintf(stdout, "Starting burn in process. Selected rwsd = %f\n", mct.rwsd);

  // initialize burn in sequence
  for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
    mclocv.current[dim_idx] = mcdata.burn[dim_idx];

  // load current samples on GPU
  hipMemcpy(dev_samples, mclocv.current, sz.samples_actual, hipMemcpyHostToDevice);
  // load kernel, calculate the current likelihood
  Likelihood_v1_sequential_addressing<<<mcin.Nd, mcin.dmap>>>>(dev_samples, dev_data, dev_labels, 
                                                               dev_lhood);
  // return back a vector of likelihoods
  hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
  // finish single result on CPU
  clhood = 0;
  for(lhood_idx = 0; lhood_idx < mcin.Nd; lhood_idx++){
    clhood += host_lhood[lhood_idx];
  }

  // calculate the current posterior
  mcloc->cposterior = log_prior(mclocv.current, mcin) + clhood;

  // start burn in
  for(i=1; i<mcin.burnin; i++)
  {
    // propose next sample
    for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] + gsl_ran_gaussian_ziggurat(r, mct.rwsd); // random walk using Marsaglia-Tsang ziggurat algorithm
  
    // load proposed samples on GPU
    hipMemcpy(dev_samples, mclocv.proposed, sz.samples_actual, hipMemcpyHostToDevice);
    // load kernel, calculate the current likelihood
    Likelihood_v1_sequential_addressing<<<mcin.Nd, mcin.dmap>>>>(dev_samples, dev_data, dev_labels, 
                                                                 dev_lhood);
    // return back a vector of likelihoods
    hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
    // finish single result on CPU
    plhood = 0;
    for(lhood_idx = 0; lhood_idx < mcin.Nd; lhood_idx++){
      plhood += host_lhood[lhood_idx];
    }
    
    mcloc->acceptance = acceptance_ratio(mclocv, mcloc, mcin, plhood);
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)
    {
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
      {
        mcdata.burn[i*mcin.ddata + dim_idx] = mclocv.proposed[dim_idx];
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior;
    }else{
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
        mcdata.burn[i*mcin.ddata + dim_idx] = mclocv.current[dim_idx];
    }
  }
  fprintf(stdout, "Burn in process finished.\n\n");
}

// tune rwsd for a target acceptance ratio
void tune_ess_gpu(gsl_rng *r, mcmc_str mcin, mcmc_tune_str mct, mcmc_v_str mcdata, 
                  mcmc_int_v mclocv, mcmc_int mcloc, sz_str sz,
                  double *dev_samples, double *dev_data, double *dev_labels,
                  double *dev_lhood, double *host_lhood)
{
  int chain_length = 5000;
  int runs = 40;
  double target_a[] = {0.10, 0.15, 0.20, 0.25, 0.30, 0.35, 0.40, 0.45, 0.50};
  double error_tolerance = 0.01;
  double min_error = 9999999;
  double max_ess = -9999999;
  double lagidx = 500;

  double sd = mct->rwsd;

  int accepted_samples, run, a_idx;
  double acc_ratio_c, acc_error_c, best_acc_ratio;
  double circ_sum, best_sd, ess_sd, ess_c;

  double *samples = NULL;
  samples = (double*) malloc(mcin.ddata * chain_length * sizeof(double));
  if(samples == NULL)
    fprintf(stderr, "ERROR: Samples vector did not allocated.\n");
  double *autocorr_lagk = NULL;
  autocorr_lagk = (double*) malloc(lagidx * sizeof(double));
  if(autocorr_lagk == NULL)
    fprintf(stderr, "ERROR: Autocorrelation vector did not allocated.\n");

  fprintf(stdout, "\nStarting tuning process. Rwsd = %5.3f\n", sd);
  
  for(a_idx=0; a_idx<9; a_idx++){
    fprintf(stdout, "\tStarting tuning for target ratio = %4.3f. Current rwsd = %5.3f\n", target_a[a_idx], sd);    
    min_error = 9999999;
    for(run=0; run<runs; run++)
    {
      fprintf(stdout, "\t\tStarting Run %2d. Current rwsd = %5.3f\n", run, sd);
      accepted_samples = 0;

      short_run_burn_in(r, mclocv, mcin, sd, &mcloc, sz,
                          dev_samples, dev_data, dev_labels,
                          dev_lhood, host_lhood);
      short_run_metropolis(r, mclocv, mcin, chain_length, sd, &mcloc, 
                            samples, &accepted_samples, sz, dev_samples, 
                            dev_data, dev_labels, dev_lhood, host_lhood);
      
      acc_ratio_c = accepted_samples/(double)chain_length;
      acc_error_c = fabs(acc_ratio_c - target_a[a_idx]);

      if(acc_error_c < min_error) // accept the current sd
      {
        best_sd = sd;
        min_error = acc_error_c;
        best_acc_ratio = acc_ratio_c;
        fprintf(stdout, "\t\t\tAccepted: rwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                        best_sd, best_acc_ratio, min_error);
      }else{
        fprintf(stdout, "\t\t\trwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                          sd, acc_ratio_c, acc_error_c);
      }
      
      if(min_error < error_tolerance) 
        break;
      
      sd *= acc_ratio_c/target_a[a_idx];
    }
    
    circ_sum = circular_autocorrelation(autocorr_lagk, samples, mcin.ddata,
                                        chain_length, lagidx);
    ess_c = chain_length / (1 + 2*circ_sum);
    
    if(ess_c > max_ess)
    {
      max_ess = ess_c;
      ess_sd = sd;
      fprintf(stdout, "\tAccepted: ess = %8.3f, rwsd = %5.3f\n", max_ess, ess_sd);
    }else{
      fprintf(stdout, "\tess= %8.3f, rwsd = %5.3f\n", ess_c, sd);
    }
  }
  mct->rwsd = ess_sd;
  fprintf(stdout, "Tuning finished. Selected rwsd = %5.3f\n\n", mct->rwsd);
  
  free(samples);
  free(autocorr_lagk);
  free_mcmc_vectors_cpu(mclocv);
}


// tune rwsd for a target acceptance ratio
void tune_target_a_gpu_v2(gsl_rng *r, mcmc_str mcin, mcmc_tune_str mct, mcmc_v_str mcdata, 
                          mcmc_int_v mclocv, mcmc_int mcloc, sz_str sz,
                          double *dev_samples, double *dev_data, double *dev_labels,
                          double *dev_lhood, double *host_lhood)
{
  int chain_length = 5000;
  int runs = 40;
  double target_a = 0.25;
  double error_tolerance = 0.01;
  double min_error = 9999999;

  double sd = mct->rwsd;

  int accepted_samples, run;
  double acc_ratio_c, acc_error_c, best_acc_ratio, best_sd;

  double *samples = NULL;
  samples = (double*) malloc(mcin.ddata * chain_length * sizeof(double));
  if(samples == NULL)
    fprintf(stderr, "ERROR: Samples vector did not allocated.\n");

  fprintf(stdout, "\nStarting tuning process. Rwsd = %5.3f\n", sd);
  
  for(run=0; run<runs; run++)
  {
    fprintf(stdout, "\tStarting Run %2d. Current rwsd = %5.3f\n", run, sd);
    accepted_samples = 0;

    short_run_burn_in(r, mclocv, mcin, sd, &mcloc, sz,
                        dev_samples, dev_data, dev_labels,
                        dev_lhood, host_lhood);
    short_run_metropolis(r, mclocv, mcin, chain_length, sd, &mcloc, 
                          samples, &accepted_samples, sz, dev_samples, 
                          dev_data, dev_labels, dev_lhood, host_lhood);

    acc_ratio_c = accepted_samples/(double)chain_length;
    acc_error_c = fabs(acc_ratio_c - target_a);

    if(acc_error_c < min_error) // accept the current sd
    {
      best_sd = sd;
      min_error = acc_error_c;
      best_acc_ratio = acc_ratio_c;
      fprintf(stdout, "\t\tAccepted: rwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                      best_sd, best_acc_ratio, min_error);
    }else{
      fprintf(stdout, "\t\trwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                        sd, acc_ratio_c, acc_error_c);
    }
    
    if(min_error < error_tolerance) 
      break;
    
    sd *= acc_ratio_c/target_a;
  }

  mct->rwsd = best_sd;
  fprintf(stdout, "Tuning finished. Selected rwsd = %5.3f\n\n", mct->rwsd);
  
  free(samples);
  free_mcmc_vectors_cpu(mclocv);
}

// tune rwsd for a target acceptance ratio
void tune_target_a_gpu(gsl_rng *r, mcmc_str mcin, mcmc_tune_str mct, mcmc_v_str mcdata, 
                        mcmc_int_v mclocv, mcmc_int mcloc, sz_str sz,
                        double *dev_samples, double *dev_data, double *dev_labels,
                        double *dev_lhood, double *host_lhood)
{
  int chain_length = 5000;
  int run = 0;
  double target_a = 0.25;
  double error_tolerance = 0.01;
  double mult_factor = 0.1;

  double sd = mct->rwsd;

  int accepted_samples;
  double acc_ratio_c;

  double *samples = NULL;
  samples = (double*) malloc(mcin.ddata * chain_length * sizeof(double));
  if(samples == NULL)
    fprintf(stderr, "Samples vector did not allocated.\n");

  fprintf(stdout, "\nStarting tuning process. Rwsd = %5.3f\n", sd);
  
  while(1)
  {
    fprintf(stdout, "\tStarting Run %2d. Current rwsd = %5.3f, Acceptance = ", run, sd);
    accepted_samples = 0;

    short_run_burn_in(r, mclocv, mcin, sd, &mcloc, sz,
                        dev_samples, dev_data, dev_labels,
                        dev_lhood, host_lhood);
    short_run_metropolis(r, mclocv, mcin, chain_length, sd, &mcloc, 
                          samples, &accepted_samples, sz, dev_samples, 
                          dev_data, dev_labels, dev_lhood, host_lhood);
    
    acc_ratio_c = accepted_samples/(double)chain_length;
    fprintf(stdout, "%4.3f\n", acc_ratio_c);

    if(acc_ratio_c > target_a + error_tolerance){
      sd *= (1+mult_factor);
    }else if(acc_ratio_c < target_a - error_tolerance){
      sd *= (1-mult_factor);
    }else{
      break;
    }
    run++;    
  }
  
  mct->rwsd = sd;
  fprintf(stdout, "Tuning finished. Selected rwsd = %5.3f\n\n", mct->rwsd);
  
  free(samples);
  free_mcmc_vectors_cpu(mclocv);
}

void short_run_burn_in(gsl_rng *r, mcmc_int_v mclocv, mcmc_str mcin, 
                        double sd, mcmc_int *mcloc, sz_str sz,
                        double *dev_samples, double *dev_data, double *dev_labels,
                        double *dev_lhood, double *host_lhood)
{
  int i, dim_idx, lhood_idx;
  double plhood, clhood;

  // initialize burn in sequence
  for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
    mclocv.current[dim_idx] = 0;
  
  // load current samples on GPU
  hipMemcpy(dev_samples, mclocv.current, sz.samples_actual, hipMemcpyHostToDevice);
  // load kernel, calculate the current likelihood
  Likelihood_v1_sequential_addressing<<<mcin.Nd, mcin.dmap>>>>(dev_samples, dev_data, dev_labels, 
                                                               dev_lhood);
  // return back a vector of likelihoods
  hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
  // finish single result on CPU
  clhood = 0;
  for(lhood_idx = 0; lhood_idx < mcin.Nd; lhood_idx++){
    clhood += host_lhood[lhood_idx];
  }

  // calculate the current posterior
  mcloc->cposterior = log_prior(mclocv.current, mcin) + clhood;  

  // start burn-in
  for(i=1; i<mcin.burnin; i++)
  {
    for(dim_idx = 0; dim_idx < mcin.ddata; dim_idx++){
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] 
                                  + gsl_ran_gaussian_ziggurat(r, sd); // random walk using Marsaglia-Tsang ziggurat algorithm  
    }

    // load proposed samples on GPU
    hipMemcpy(dev_samples, mclocv.proposed, sz.samples_actual, hipMemcpyHostToDevice);
    // load kernel, calculate the current likelihood
    Likelihood_v1_sequential_addressing<<<mcin.Nd, mcin.dmap>>>>(dev_samples, dev_data, dev_labels, 
                                                                 dev_lhood);
    // return back a vector of likelihoods
    hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
    // finish single result on CPU
    plhood = 0;
    for(lhood_idx = 0; lhood_idx < mcin.Nd; lhood_idx++){
      plhood += host_lhood[lhood_idx];
    }
    
    mcloc->acceptance = acceptance_ratio(mclocv, mcloc, mcin, plhood);
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)    // decide if you accept the proposed theta or not
    {
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior; // make proposed posterior the current 
    }
  }
}

void short_run_metropolis(data_str data, gsl_rng *r, mcmc_int_v mclocv, mcmc_str mcin, 
                          int chain_length, double sd, mcmc_int *mcloc, double *samples, 
                          int *accepted_samples, sz_str sz, double *dev_samples, 
                          double *dev_data, double *dev_labels,
                          double *dev_lhood, double *host_lhood)
{
  int i, dim_idx;
  int lhood_idx;
  double plhood;
  
  // start metropolis
  for(i=0; i < chain_length; i++){    
    for(dim_idx = 0; dim_idx < mcin.ddata; dim_idx++){
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] 
                                  + gsl_ran_gaussian_ziggurat(r, sd); // random walk using Marsaglia-Tsang ziggurat algorithm    
    }

    // load proposed samples on GPU
    hipMemcpy(dev_samples, mclocv.proposed, sz.samples_actual, hipMemcpyHostToDevice);
    // load kernel, calculate the proposed likelihood
    Likelihood_v1_sequential_addressing<<<mcin.Nd, mcin.dmap>>>>(dev_samples, dev_data, dev_labels, 
                                                                 dev_lhood);
    // return back a vector of likelihoods
    hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
    // finish single result on CPU
    plhood = 0;
    for(lhood_idx = 0; lhood_idx < mcin.Nd; lhood_idx++){
      plhood += host_lhood[lhood_idx];
    }

    // calculate acceptance ratio
    mcloc->acceptance = acceptance_ratio(mclocv, mcloc, mcin, plhood);
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)    // decide if you accept the proposed theta or not
    {
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
        samples[i*mcin.ddata + dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior; // make proposed posterior the current 
      *accepted_samples += 1; 
    }else{
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){
        samples[i*mcin.ddata + dim_idx] = mclocv.current[dim_idx];
      }      
    }     
  }
}

double acceptance_ratio(mcmc_int_v mclocv, mcmc_int *mcloc, mcmc_str mcin, double plhood) 
{
  double log_ratio;
  mcloc->pposterior = log_prior(mclocv.proposed, mcin) + plhood;
  log_ratio = mcloc->pposterior - mcloc->cposterior;

  return exp(log_ratio);
}

double log_prior(double *sample, mcmc_str mcin)
{ 
  double log_prob = 0;
  int dim_idx;

  for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){  //assuming iid priors
    log_prob += log(gsl_ran_gaussian_pdf(sample[dim_idx], PRIOR_SD));
  }

  return log_prob;
}

void print_gpu_info()
{
  hipDeviceProp_t  prop;
  int count;
  hipGetDeviceCount( &count );

  for (int i=0; i< count; i++) {
    hipGetDeviceProperties( &prop, i );
    //Do something with our device's properties
    printf( " --- General Information for device %d ---\n", i ); 
    printf( "Name: %s\n", prop.name );
    printf( "Compute capability: %d.%d\n", prop.major, prop.minor ); 
    printf( "Clock rate: %d\n", prop.clockRate );
    printf( "Device copy overlap: " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" ); 
    else
      printf( "Disabled\n" );

    printf( "Kernel execition timeout : " ); 
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" ); 
    else
      printf( "Disabled\n" );

    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
             prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
               prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1],
               prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
               prop.maxGridSize[0], prop.maxGridSize[1],
               prop.maxGridSize[2] );
    printf( "\n" );
  }  
}

/****************************************************************************/
void Metropolis_Hastings_gpu(data_str data, gsl_rng *r, mcmc_str mcin,
                              mcmc_tune_str mct, mcmc_v_str mcdata,
                              out_str *out_par)
{
  print_gpu_info();
  hipSetDevice(0);

  mcmc_int_v mclocv;
  mcmc_int mcloc;
  malloc_mcmc_vectors_gpu(&mclocv, mcin);

  int accepted_samples = 0;
  int dim_idx,i;

  int threadsPerBlock = mcin.dmap;
  int blocksPerGrid = mcin.Nd;

  sz_str sz;
  sz.samples = mcin.dmap * sizeof(double);
  sz.data = mcin.dmap * mcin.Nd * sizeof(double);
  sz.labels = mcin.Nd * sizeof(int8_t);
  sz.likelihood = blocksPerGrid * sizeof(double);

  double *host_lhood = (double *) malloc(sz.likelihood * sizeof(double));
  double *dev_samples, *dev_data, *dev_lhood;
  int8_t *dev_labels;
  hipMalloc(&dev_samples, sz.samples);
  hipMalloc(&dev_data, sz.data);
  hipMalloc(&dev_labels, sz.labels);
  hipMalloc(&dev_lhood, sz.likelihood);    // kernel will return a vector of likelihoods

  clock_t start, stop;
  start  = clock();

  // initialisation
  memcpy(mclocv.current, mcdata.burn, mcin.ddata*sizeof(double));

  // load data, labels and current samples on gpu
  hipMemcpy(dev_data, data.gpudata, sz.data, hipMemcpyHostToDevice);
  hipMemcpy(dev_labels, data.labels, sz.labels, hipMemcpyHostToDevice);
  
  hipMemcpy(dev_samples, mclocv.current, sz.samples, hipMemcpyHostToDevice);
  Likelihood_v1<<<blocksPerGrid, threadsPerBlock>>>(dev_samples, dev_data, dev_labels, 
                                                    mcin.ddata, mcin.dmap, mcin.Nd, 
                                                    dev_lhood);
  hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
  // finish on cpu
  uint like_idx;
  double cpu_likelihood = 0;
  for(like_idx = 0; like_idx < sz.likelihood; like_idx++){
    cpu_likelihood += host_lhood[like_idx];
  }

  mcloc.cposterior = log_prior(mclocv.current, mcin) + cpu_likelihood;
  
  //perfrom metropolis-hastings algorithm
  for(i=1; i<(mcin.Ns+mcin.burnin); i++){  
    for(dim_idx = 0; dim_idx < mcin.ddata; dim_idx++){
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] 
                                  + gsl_ran_gaussian_ziggurat(r, mct.rwsd); // random walk using Marsaglia-Tsang ziggurat algorithm
    }

    hipMemcpy(dev_samples, mclocv.current, sz.samples, hipMemcpyHostToDevice);
    Likelihood_v1<<<blocksPerGrid, threadsPerBlock>>>(dev_samples, dev_data, dev_labels, 
                                                      mcin.ddata, mcin.dmap, mcin.Nd, 
                                                      dev_lhood);
    hipMemcpy(host_lhood, dev_lhood, sz.likelihood, hipMemcpyDeviceToHost);
    // finish on cpu
    cpu_likelihood = 0;
    for(like_idx = 0; like_idx < sz.likelihood; like_idx++){
      cpu_likelihood += host_lhood[like_idx];
    }

    mcloc.acceptance = acceptance_ratio(mclocv, &mcloc, mcin, cpu_likelihood);// Calculate acceptance ratio in the log domain
    mcloc.u = gsl_rng_uniform(r);

    if(mcloc.u <= mcloc.acceptance)    // decide if you accept the proposed theta or not
    {
      memcpy(mclocv.temp, mclocv.proposed, mcin.ddata*sizeof(double));
      memcpy(mclocv.current, mclocv.proposed, mcin.dmap*sizeof(double));
      mcloc.cposterior = mcloc.pposterior; // make proposed posterior the current
      accepted_samples += 1;   
    }else{
      memcpy(mclocv.temp, mclocv.current, mcin.ddata*sizeof(double));
    }

    if(i < mcin.burnin){
      for(dim_idx = 0; dim_idx < mcin.ddata; dim_idx++)
        mcdata.burn[(i * mcin.ddata) + dim_idx] = mclocv.temp[dim_idx];
    }else{
      for(dim_idx = 0; dim_idx < mcin.ddata; dim_idx++)
        mcdata.samples[(i - mcin.burnin) * mcin.ddata + dim_idx] = mclocv.temp[dim_idx];
    }        
  }

  stop = clock() - start;
  out_par->time_m = stop / (CLOCKS_PER_SEC * 60);
  out_par->time_s = (stop / CLOCKS_PER_SEC) - (out_par->time_m * 60);
  out_par->time_ms = (stop * 1000 / CLOCKS_PER_SEC) - (out_par->time_s * 1000) 
                      - (out_par->time_m * 1000 * 60);

  out_par -> acc_ratio = (double)accepted_samples / mcin.Ns;

  free_mcmc_vectors_gpu(mclocv);
  free(host_lhood);
}


