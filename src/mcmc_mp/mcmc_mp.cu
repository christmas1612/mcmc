#include "hip/hip_runtime.h"
#ifndef __MCMC_MP_CU__
#define __MCMC_MP_CU__

#include "mcmc_mp.h"

const int PRIOR_SD = 5;

void mp_sampler(data_str data, gsl_rng *r, mcmc_str mcin,
                  mcmc_tune_str *mct, mcmc_v_str mcdata,
                  gpu_v_str gpu, out_str *res)
{
  int accepted_samples;
  clock_t startTune, stopTune;
  clock_t startBurn, stopBurn;
  clock_t startMcmc, stopMcmc;
  // print_gpu_info();
  hipSetDevice(0);

  hipblasHandle_t handle;

  mcmc_int_v mclocv;
  mcmc_int mcloc;
  mcloc.cposterior = 0;
  mcloc.pposterior = 0;
  mcloc.acceptance = 0;
  mcloc.u = 0;
  malloc_mcmc_vectors(&mclocv, mcin);

  // set up the gpu vectors
  sz_str sz;
  dev_v_str d;

  getBlocksAndThreads(gpu.kernel, mcin.Nd, gpu.maxBlocks, gpu.maxThreads, &gpu.blocks, &gpu.threads);

  getSizes_mp(&sz, mcin, gpu);
  
  hipMalloc(&d.samples, sz.samples);           hipMalloc(&d.samplesf, sz.samplesf);
  hipMalloc(&d.data, sz.data);                 hipMalloc(&d.dataf, sz.dataf);
  hipMalloc(&d.cuLhood, sz.cuLhood);           hipMalloc(&d.cuLhoodf, sz.cuLhoodf);
  hipMalloc(&d.zlabels, sz.zlabels);           hipMalloc(&d.zidx, sz.zidx);
  hipMalloc(&d.brightLhood, sz.brightLhood);   hipMalloc(&d.darkLhood, sz.darkLhood);
  hipMalloc(&d.resample, sz.resample)
  hipMalloc(&d.lhood, sz.lhood);

  hipblasCreate(&handle);

  hipMemcpy(d.data, data.data, sz.data, hipMemcpyHostToDevice);
  hipMemcpy(d.dataf, data.dataf, sz.dataf, hipMemcpyHostToDevice);
  hipMemcpy(d.zlabels, data.zlabels, sz.zlabels, hipMemcpyHostToDevice);
  hipMemcpy(d.zidx, data.zidx, sz.zidx, hipMemcpyHostToDevice);

  startTune = clock();
  if(mcin.tune == 1)
    tune_target_a_gpu_v2(handle, r, mcin, mct, mclocv, mcloc, sz, gpu, d, data.mvout);
  else if(mcin.tune == 2)  
    tune_ess_gpu(handle, r, mcin, mct, mclocv, mcloc, sz, gpu, d, data.mvout);    
  stopTune = clock() - startTune;

  startBurn = clock();
  if(mcin.burnin != 0)
    burn_in_metropolis_mp(handle, r, mcin, mct, mcdata, mclocv, &mcloc, sz, gpu, d, data.mvout);
  stopBurn = clock() - startBurn;

  accepted_samples = 0;  

  startMcmc = clock();
  metropolis_gpu(handle, r, mcin, mct, mcdata, mclocv, &mcloc, &accepted_samples, sz, gpu, d, data.mvout, res);
  stopMcmc = clock() - startMcmc;

  res->tuneTime = stopTune * 1000 / CLOCKS_PER_SEC;   // tuning time in ms
  res->burnTime = stopBurn * 1000 / CLOCKS_PER_SEC;   // burn in time in ms
  res->mcmcTime = stopMcmc * 1000 / CLOCKS_PER_SEC;   // mcmc time in ms
  res->acceptance = (double)accepted_samples / mcin.Ns;
  
  hipFree(d.samples);      hipFree(d.samplesf);
  hipFree(d.data);         hipFree(d.dataf);
  hipFree(d.cuLhood);      hipFree(d.cuLhoodf);
  hipFree(d.zlabels);      hipFree(d.zidx);
  hipFree(d.brightLhood);  hipFree(d.darkLhood);
  hipFree(d.resample);
  hipFree(d.lhood);   

  free_mcmc_vectors(mclocv, mcin);
  hipblasDestroy(handle);
}

void metropolis_mp(hipblasHandle_t handle, gsl_rng *r, mcmc_str mcin,
                    mcmc_tune_str *mct, mcmc_v_str mcdata, mcmc_int_v mclocv, 
                    mcmc_int *mcloc, int *accepted_samples, sz_str sz,
                    gpu_v_str gpu, dev_v_str d, double *host_lhood, out_str *res)
{
  int i, dim_idx;
  double plhood;
  res->cuTime = 0;
  res->cuBandwidth = 0;
  res->kernelTime = 0;
  res->kernelBandwidth = 0;
  res->gpuTime = 0;
  res->gpuBandwidth = 0;

  fprintf(stdout, "Starting metropolis algorithm. Selected rwsd = %f\n", mct->rwsd); 
  
  for(i=0; i<mcin.Ns; i++)
  {
    for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
    {
      // random walk using Marsaglia-Tsang ziggurat algorithm
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] + gsl_ran_gaussian_ziggurat(r, mct->rwsd);
      mclocv.proposedf[dim_idx] = mclocv.proposed[dim_idx];
    }

    plhood = mp_likelihood(handle, mcin, gpu, mclocv.proposed, sz.samples, d, host_lhood, res);
    
    // calculate acceptance ratio
    mcloc->acceptance = acceptance_ratio_mp(mclocv, mcloc, mcin, plhood);
    
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)
    {
      // accept proposed sample
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
      {
        mcdata.samples[i*mcin.ddata + dim_idx] = mclocv.proposed[dim_idx];
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior;
      *accepted_samples += 1;
    }else{
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
        mcdata.samples[i*mcin.ddata + dim_idx] = mclocv.current[dim_idx];
    }    
  } 
  fprintf(stdout, "Metropolis algorithm finished. Accepted Samples = %d\n\n", *accepted_samples);
}

void burn_in_metropolis_mp(hipblasHandle_t handle, gsl_rng *r, mcmc_str mcin, 
                            mcmc_tune_str *mct, mcmc_v_str mcdata, 
                            mcmc_int_v mclocv, mcmc_int *mcloc, sz_str sz,
                            gpu_v_str gpu, dev_v_str d, double *host_lhood)
{
  int i, dim_idx;
  double plhood, clhood;
  out_str res;

  fprintf(stdout, "Starting burn in process. Selected rwsd = %f\n", mct->rwsd);
  // initialize burn in sequence
  for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
  {
    mclocv.current[dim_idx] = mcdata.burn[dim_idx];
    mclocv.currentf[dim_idx] = mclocv.current[dim_idx];
  }

  clhood = mp_burnIn_likelihood(handle, mcin, gpu, mclocv.current, sz.samples, d, host_lhood, &res);
  // calculate the current posterior
  mcloc->cposterior = log_prior_mp(mclocv.current, mcin) + clhood;

  // start burn in
  for(i=1; i<mcin.burnin; i++)
  {
    // propose next sample
    for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] + gsl_ran_gaussian_ziggurat(r, mct->rwsd); // random walk using Marsaglia-Tsang ziggurat algorithm
      mclocv.proposedf[dim_idx] = mclocv.proposed[dim_idx];
    }

    plhood = mp_likelihood(handle, mcin, gpu, mclocv.proposed, sz.samples, d, host_lhood, &res);

    mcloc->acceptance = acceptance_ratio_mp(mclocv, mcloc, mcin, plhood);
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)
    {
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
      {
        mcdata.burn[i*mcin.ddata + dim_idx] = mclocv.proposed[dim_idx];
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior;
    }else{
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
        mcdata.burn[i*mcin.ddata + dim_idx] = mclocv.current[dim_idx];
    }
  }
  fprintf(stdout, "Burn in process finished.\n\n");
}


double reduction_mp_d(gpu_v_str gpu, dev_v_str d, double *host_lhood, double *ke_acc_Bytes)
{
  double gpu_result = 1;
  int i;
  int numBlocks = gpu.blocks;
  int threads, blocks;

  *ke_acc_Bytes = gpu.size * sizeof(double);

  reduceSum_d(gpu.size, gpu.threads, gpu.blocks, gpu.kernel, d.cuLhood, d.brightLhood, 2);

  while(numBlocks >= gpu.cpuThresh)
  {
    getBlocksAndThreads(gpu.kernel, numBlocks, gpu.maxBlocks, gpu.maxThreads, &blocks, &threads);
    
    ke_acc_Bytes += numBlocks * sizeof(double);
    
    hipMemcpy(d.cuLhood, d.brightLhood, numBlocks*sizeof(double), hipMemcpyDeviceToDevice);
    reduceSum_d(numBlocks, threads, blocks, gpu.kernel, d.cuLhood, d.lhood, 2);    

    if(gpu.kernel < 3)
    {
      numBlocks = (numBlocks + threads - 1) / threads;
    }else{
      numBlocks = (numBlocks +(threads*2-1)) / (threads*2);
    }
  }

  hipMemcpy(host_lhood, d.brightLhood, numBlocks*sizeof(double), hipMemcpyDeviceToHost);  
  // accumulate result on CPU
  for(i=0; i<numBlocks; i++){
    gpu_result *= host_lhood[i];
  }

  return gpu_result;
}

float reduction_mp_f(gpu_v_str gpu, dev_v_str d, float *host_lhood, double *ke_acc_Bytes)
{
  double gpu_result = 1;
  int i;
  int numBlocks = gpu.blocks;
  int threads, blocks;

  *ke_acc_Bytes = gpu.size * sizeof(float);

  reduceSum_f(gpu.size, gpu.threads, gpu.blocks, gpu.kernel, d.cuLhood, d.darkLhood, 2);

  while(numBlocks >= gpu.cpuThresh)
  {
    getBlocksAndThreads(gpu.kernel, numBlocks, gpu.maxBlocks, gpu.maxThreads, &blocks, &threads);
    
    ke_acc_Bytes += numBlocks * sizeof(float);
    
    hipMemcpy(d.cuLhoodf, d.darkLhood, numBlocks*sizeof(float), hipMemcpyDeviceToDevice);
    reduceSum_f(numBlocks, threads, blocks, gpu.kernel, d.cuLhoodf, d.darkLhood, 2);    

    if(gpu.kernel < 3)
    {
      numBlocks = (numBlocks + threads - 1) / threads;
    }else{
      numBlocks = (numBlocks +(threads*2-1)) / (threads*2);
    }
  }

  hipMemcpy(host_lhood, d.darkLhood, numBlocks*sizeof(float), hipMemcpyDeviceToHost);  
  // accumulate result on CPU
  for(i=0; i<numBlocks; i++){
    gpu_result *= host_lhood[i];
  }

  return gpu_result;
}

double gpu_likelihood_d(hipblasHandle_t handle, mcmc_str mcin, gpu_v_str gpu,
                        double *samples, size_t sampleSz, 
                        float *samplesf, size_t sampleSzf, dev_v_str d, 
                        double *host_lhood, out_str *res)
{
  double ke_acc_Bytes = 0;
  double cuBytes = 0;
  double red_d = 0;
  float red_f = 0;
  double a = 1.0;
  double b = 0.0;
  float cu_ms = 0;
  float ke_ms = 0;
  double mp_lhood = 0;

  hipEvent_t cuStart, cuStop, keStart, keStop;
  hipEventCreate(&cuStart); 
  hipEventCreate(&cuStop);
  hipEventCreate(&keStart);
  hipEventCreate(&keStop);  

  hipMemcpy(d.samples, samples, sampleSz, hipMemcpyHostToDevice);
  hipMemcpy(d.samplesf, samplesf, sampleSzf, hipMemcpyHostToDevice);

  hipEventRecord(cuStart);  

  getBlocksAndThreads(gpu.kernel, mcin.bright+mcin.cand, gpu.maxBlocks, gpu.maxThreads, &blocks, &threads);
  brightL(threads, blocks, d, mcin);
  hipDeviceSynchronize();

  getBlocksAndThreads(gpu.kernel, mcin.dark, gpu.maxBlocks, gpu.maxThreads, &blocks, &threads);
  darkL(threads, blocks, d, mcin);
  hipDeviceSynchronize();

  hipEventRecord(cuStop);

  hipEventRecord(keStart);

  getBlocksAndThreads(gpu.kernel, mcin.bright, gpu.maxBlocks, gpu.maxThreads, &blocks, &threads);
  red_d = reduction_mp_d(gpu, d, host_lhood, &ke_acc_Bytes);
  hipDeviceSynchronize();

  getBlocksAndThreads(gpu.kernel, mcin.dark+mcin.cand, gpu.maxBlocks, gpu.maxThreads, &blocks, &threads);
  red_f = reduction_mp_f(gpu, d, host_lhood, &ke_acc_Bytes);
  hipDeviceSynchronize();
  mp_lhood = log(red_d) + log(red_f);
  hipEventRecord(keStop);

  // resample

  hipEventSynchronize(cuStop); 
  hipEventSynchronize(keStop);
  hipEventElapsedTime(&cu_ms, cuStart, cuStop);
  hipEventElapsedTime(&ke_ms, keStart, keStop);

  cuBytes = mcin.Nd * (mcin.ddata + 2) * sizeof(double);

  res->cuTime += cu_ms / mcin.Ns;    // average cuBlas time
  res->cuBandwidth += (cuBytes / cu_ms / 1e6) / mcin.Ns;
  res->kernelTime += ke_ms / mcin.Ns;
  res->kernelBandwidth += (ke_acc_Bytes / ke_ms / 1e6) / mcin.Ns;
  res->gpuTime += (cu_ms + ke_ms) / mcin.Ns;
  res->gpuBandwidth += ((cuBytes + ke_acc_Bytes) / (cu_ms + ke_ms) / 1e6) / mcin.Ns;

  return mp_lhood;  
}

// // tune rwsd for a target acceptance ratio
void tune_ess_gpu(hipblasHandle_t handle, gsl_rng *r, mcmc_str mcin, mcmc_tune_str *mct, 
                  mcmc_int_v mclocv, mcmc_int mcloc, sz_str sz,
                  gpu_v_str gpu, dev_v_str d, double *host_lhood)
{
  int chain_length = 5000;
  int runs = 40;
  double target_a[] = {0.10, 0.15, 0.20, 0.25, 0.30, 0.35, 0.40, 0.45, 0.50};
  double error_tolerance = 0.01;
  double min_error = 9999999;
  double max_ess = -9999999;
  double lagidx = 500;

  double sd = mct->rwsd;
  double ess_sd = sd;

  int accepted_samples, run, a_idx;
  double acc_ratio_c, acc_error_c, best_acc_ratio;
  double circ_sum, best_sd, ess_c;

  double *samples = NULL;
  samples = (double*) malloc(mcin.ddata * chain_length * sizeof(double));
  if(samples == NULL)
    fprintf(stderr, "ERROR: Samples vector did not allocated.\n");
  double *autocorr_lagk = NULL;
  autocorr_lagk = (double*) malloc(lagidx * sizeof(double));
  if(autocorr_lagk == NULL)
    fprintf(stderr, "ERROR: Autocorrelation vector did not allocated.\n");

  fprintf(stdout, "\nStarting tuning process. Rwsd = %5.3f\n", sd);
  
  for(a_idx=0; a_idx<9; a_idx++){
    fprintf(stdout, "\tStarting tuning for target ratio = %4.3f. Current rwsd = %5.3f\n", target_a[a_idx], sd);    
    min_error = 9999999;
    for(run=0; run<runs; run++)
    {
      fprintf(stdout, "\t\tStarting Run %2d. Current rwsd = %5.3f\n", run, sd);
      accepted_samples = 0;

      short_run_burn_in_gpu(handle, r, mclocv, mcin, sd, &mcloc, sz, gpu, d, host_lhood);
      short_run_metropolis_gpu(handle, r, mclocv, mcin, chain_length, sd, &mcloc, 
                                samples, &accepted_samples, sz, gpu, d, host_lhood);
      
      acc_ratio_c = accepted_samples/(double)chain_length;
      acc_error_c = fabs(acc_ratio_c - target_a[a_idx]);

      if(acc_error_c < min_error) // accept the current sd
      {
        best_sd = sd;
        min_error = acc_error_c;
        best_acc_ratio = acc_ratio_c;
        fprintf(stdout, "\t\t\tAccepted: rwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                        best_sd, best_acc_ratio, min_error);
      }else{
        fprintf(stdout, "\t\t\trwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                          sd, acc_ratio_c, acc_error_c);
      }
      
      if(min_error < error_tolerance) 
        break;
      
      sd *= acc_ratio_c/target_a[a_idx];
    }
    
    circ_sum = circular_autocorrelation(autocorr_lagk, samples, mcin.ddata,
                                        chain_length, lagidx);
    ess_c = chain_length / (1 + 2*circ_sum);
    
    if(ess_c > max_ess)
    {
      max_ess = ess_c;
      ess_sd = sd;
      fprintf(stdout, "\tAccepted: ess = %8.3f, rwsd = %5.3f\n", max_ess, ess_sd);
    }else{
      fprintf(stdout, "\tess= %8.3f, rwsd = %5.3f\n", ess_c, sd);
    }
  }
  mct->rwsd = ess_sd;
  fprintf(stdout, "Tuning finished. Selected rwsd = %5.3f\n\n", mct->rwsd);
  
  free(samples);
  free(autocorr_lagk);
}


// // tune rwsd for a target acceptance ratio
void tune_target_a_gpu_v2(hipblasHandle_t handle, gsl_rng *r, mcmc_str mcin, mcmc_tune_str *mct, 
                          mcmc_int_v mclocv, mcmc_int mcloc, sz_str sz, gpu_v_str gpu, dev_v_str d, 
                          double *host_lhood)
{
  int chain_length = 5000;
  int runs = 40;
  double target_a = 0.25;
  double error_tolerance = 0.01;
  double min_error = 9999999;

  double sd = mct->rwsd;
  double best_sd = sd;
  int accepted_samples, run;
  double acc_ratio_c, acc_error_c, best_acc_ratio;

  double *samples = NULL;
  samples = (double*) malloc(mcin.ddata * chain_length * sizeof(double));
  if(samples == NULL)
    fprintf(stderr, "ERROR: Samples vector did not allocated.\n");

  fprintf(stdout, "\nStarting tuning process. Rwsd = %5.3f\n", sd);
  
  for(run=0; run<runs; run++)
  {
    fprintf(stdout, "\tStarting Run %2d. Current rwsd = %5.3f\n", run, sd);
    accepted_samples = 0;

    short_run_burn_in_gpu(handle, r, mclocv, mcin, sd, &mcloc, sz, gpu, d, host_lhood);
    short_run_metropolis_gpu(handle, r, mclocv, mcin, chain_length, sd, &mcloc, 
                              samples, &accepted_samples, sz, gpu, d, host_lhood);

    acc_ratio_c = accepted_samples/(double)chain_length;
    acc_error_c = fabs(acc_ratio_c - target_a);

    if(acc_error_c < min_error) // accept the current sd
    {
      best_sd = sd;
      min_error = acc_error_c;
      best_acc_ratio = acc_ratio_c;
      fprintf(stdout, "\t\tAccepted: rwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                      best_sd, best_acc_ratio, min_error);
    }else{
      fprintf(stdout, "\t\trwsd = %5.3f, acceptance = %4.3f, error = %4.3f\n", 
                        sd, acc_ratio_c, acc_error_c);
    }
    
    if(min_error < error_tolerance) 
      break;
    
    sd *= acc_ratio_c/target_a;
  }

  mct->rwsd = best_sd;
  fprintf(stdout, "Tuning finished. Selected rwsd = %5.3f\n\n", mct->rwsd);
  
  free(samples);
}

// tune rwsd for a target acceptance ratio
void tune_target_a_gpu(hipblasHandle_t handle, gsl_rng *r, mcmc_str mcin, mcmc_tune_str *mct, 
                        mcmc_int_v mclocv, mcmc_int mcloc, sz_str sz, gpu_v_str gpu, dev_v_str d, 
                        double *host_lhood)
{
  int chain_length = 5000;
  int run = 0;
  double target_a = 0.25;
  double error_tolerance = 0.01;
  double mult_factor = 0.1;

  double sd = mct->rwsd;

  int accepted_samples;
  double acc_ratio_c;

  double *samples = NULL;
  samples = (double*) malloc(mcin.ddata * chain_length * sizeof(double));
  if(samples == NULL)
    fprintf(stderr, "Samples vector did not allocated.\n");

  fprintf(stdout, "\nStarting tuning process. Rwsd = %5.3f\n", sd);
  
  while(1)
  {
    fprintf(stdout, "\tStarting Run %2d. Current rwsd = %5.3f, Acceptance = ", run, sd);
    accepted_samples = 0;

    short_run_burn_in_gpu(handle, r, mclocv, mcin, sd, &mcloc, sz, gpu, d, host_lhood);
    short_run_metropolis_gpu(handle, r, mclocv, mcin, chain_length, sd, &mcloc, 
                              samples, &accepted_samples, sz, gpu, d, host_lhood);
    
    acc_ratio_c = accepted_samples/(double)chain_length;
    fprintf(stdout, "%4.3f\n", acc_ratio_c);

    if(acc_ratio_c > target_a + error_tolerance){
      sd *= (1+mult_factor);
    }else if(acc_ratio_c < target_a - error_tolerance){
      sd *= (1-mult_factor);
    }else{
      break;
    }
    run++;    
  }
  
  mct->rwsd = sd;
  fprintf(stdout, "Tuning finished. Selected rwsd = %5.3f\n\n", mct->rwsd);
  
  free(samples);
}

void short_run_burn_in_gpu(hipblasHandle_t handle, gsl_rng *r, mcmc_int_v mclocv, mcmc_str mcin, 
                        double sd, mcmc_int *mcloc, sz_str sz, gpu_v_str gpu, dev_v_str d, 
                        double *host_lhood)
{
  int i, dim_idx;
  double plhood, clhood;

  out_str res;

  // initialize burn in sequence
  for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++)
    mclocv.current[dim_idx] = 0;

  clhood = gpu_likelihood_d(handle, mcin, gpu, mclocv.current, sz.samples, d, host_lhood, &res); 
  // calculate the current posterior
  mcloc->cposterior = log_prior_gpu(mclocv.current, mcin) + clhood;

  // start burn-in
  for(i=1; i<mcin.burnin; i++)
  {
    for(dim_idx = 0; dim_idx < mcin.ddata; dim_idx++){
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] 
                                  + gsl_ran_gaussian_ziggurat(r, sd); // random walk using Marsaglia-Tsang ziggurat algorithm  
    }

    plhood = gpu_likelihood_d(handle, mcin, gpu, mclocv.proposed, sz.samples, d, host_lhood, &res);

    mcloc->acceptance = acceptance_ratio_gpu(mclocv, mcloc, mcin, plhood);
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)    // decide if you accept the proposed theta or not
    {
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior; // make proposed posterior the current 
    }
  }
}

void short_run_metropolis_gpu(hipblasHandle_t handle, gsl_rng *r, mcmc_int_v mclocv, mcmc_str mcin, 
                          int chain_length, double sd, mcmc_int *mcloc, double *samples, 
                          int *accepted_samples, sz_str sz, gpu_v_str gpu, dev_v_str d, double *host_lhood)
{
  int i, dim_idx;
  double plhood;
  
  out_str res;

  // start metropolis
  for(i=0; i < chain_length; i++){    
    for(dim_idx = 0; dim_idx < mcin.ddata; dim_idx++){
      mclocv.proposed[dim_idx] = mclocv.current[dim_idx] 
                                  + gsl_ran_gaussian_ziggurat(r, sd); // random walk using Marsaglia-Tsang ziggurat algorithm    
    }

    plhood = gpu_likelihood_d(handle, mcin, gpu, mclocv.proposed, sz.samples, d, host_lhood, &res);

    mcloc->acceptance = acceptance_ratio_gpu(mclocv, mcloc, mcin, plhood);
    mcloc->u = gsl_rng_uniform(r);

    if(mcloc->u <= mcloc->acceptance)    // decide if you accept the proposed theta or not
    {
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){
        mclocv.current[dim_idx] = mclocv.proposed[dim_idx];
        samples[i*mcin.ddata + dim_idx] = mclocv.proposed[dim_idx];
      }
      mcloc->cposterior = mcloc->pposterior; // make proposed posterior the current 
      *accepted_samples += 1; 
    }else{
      for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){
        samples[i*mcin.ddata + dim_idx] = mclocv.current[dim_idx];
      }      
    }     
  }
}

double acceptance_ratio_gpu(mcmc_int_v mclocv, mcmc_int *mcloc, mcmc_str mcin, double plhood) 
{
  double log_ratio;
  mcloc->pposterior = log_prior_gpu(mclocv.proposed, mcin) + plhood;
  log_ratio = mcloc->pposterior - mcloc->cposterior;

  return exp(log_ratio);
}

double log_prior_gpu(double *sample, mcmc_str mcin)
{ 
  double log_prob = 0;
  int dim_idx;

  for(dim_idx=0; dim_idx<mcin.ddata; dim_idx++){  //assuming iid priors
    log_prob += log(gsl_ran_gaussian_pdf(sample[dim_idx], PRIOR_SD));
  }

  return log_prob;
}




#endif // __MCMC_MP_CU__